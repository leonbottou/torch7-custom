#include "hip/hip_runtime.h"
#ifndef assert
#define assert(e)  \
    if (!(e)) { \
        printf("failed assertion `%s'\n", #e); \
        THError("aborting..."); \
    };
#endif

#ifndef MIN
#define MIN(X,Y) ((X) < (Y) ? (X) : (Y))
#endif

#ifndef MAX
#define MAX(X,Y) ((X) < (Y) ? (Y) : (X))
#endif




__global__ void transpose12(float* in, float* out, int instr0, int instr1, int outstr0, int outstr1)
{
   /*
      blockIdx.x =  [ 0, kH-1 ]
      blockIdx.y =  [ 0, op-1 ]
      threadIdx.x = [ 0, 31   ]
   */
   
   in  +=   blockIdx.x * instr1 + blockIdx.y * instr0;
   out +=   blockIdx.x * outstr0 + blockIdx.y * outstr1;
   
   for (int i=threadIdx.x; i<instr1; i+=blockDim.x)
   {
      out[i]=in[i];
   }
}



void transposeWeightMatrix(THCudaTensor* in, THCudaTensor* out)
{

// this function transposes dimensions 1 and 2 of the tensor "in" and stores it in tensor "out"
// works only for 4D tensors

  int t_kH = in->size[1];
  int t_op = in->size[0];
  int t_kW = in->size[2];
  int t_ip = in->size[3];
  THCudaTensor_resize4d(out, t_kH, t_op, t_kW, t_ip);

  float* w0ptr = THCudaTensor_data(in);
  float* w1ptr = THCudaTensor_data(out);
  
  int w0str0   = in->stride[0];
  int w0str1   = in->stride[1];
  int w1str0   = out->stride[0];
  int w1str1   = out->stride[1];
  
  dim3 transposeblocks(t_kH, t_op);
  dim3 transposethreads(32);
  
  transpose12<<<transposeblocks,transposethreads>>>(w0ptr, w1ptr, w0str0, w0str1, w1str0, w1str1);

}








__global__ void SCinputcopykernelsmall(float* inputptr, float* icopyptr, int stridey, int bs, int ih, 
      int iw, int ip, int padtop, int padleft, int toh, int tiw)
{
      /* blockIdx.z  = s     [ 0, stridey-1 ]
         blockIdx.y  = it1   [ 0, bs-1      ]
         blockIdx.x  = it3   [ 0, (iw/blockDim.y)-1+1      ]
         threadIdx.x = it4x  [ 0, ip-1      ]
         threadIdx.y = it4y  [ 0, 32/ip-1   ]
      // this is the special case where ip < 32 and the input is contiguous (optimized coalescing for input layer)
       */
         
      int fout = (MAX(0,padtop-blockIdx.z)+stridey-1)/stridey;
      int fin = fout * stridey - padtop + blockIdx.z;

      if (fin < ih) 
      {
         inputptr += (blockIdx.y)*ih*iw*ip+fin*iw*ip;
         icopyptr += blockIdx.z*bs*toh*tiw*ip+(blockIdx.y)*toh*tiw*ip+fout*tiw*ip+padleft*ip;
         
         int inputsize2   = ((ih-fin) + stridey - 1) / stridey;

         for (int it2=0; it2<inputsize2; it2++) { 
            if((blockIdx.x*blockDim.y)*ip+threadIdx.x+blockDim.x*threadIdx.y<ip*iw) {
            icopyptr[(blockIdx.x*blockDim.y)*ip+threadIdx.x+blockDim.x*threadIdx.y]=inputptr[(blockIdx.x*blockDim.y)*ip+threadIdx.x+blockDim.x*threadIdx.y];
            }
            inputptr += stridey*iw*ip;
            icopyptr += tiw*ip;
			}
      }
}
      





__global__ void SCinputcopykernel(float* inputptr, float* icopyptr, int stridey, int bs, int ih, 
      int iw, int ip, int padtop, int padleft, int toh, int tiw, int inputstr0, int inputstr1, int inputstr2, int inputstr3)
{
      // blockIdx.z  = s     [ 0, stridey-1 ]
      // blockIdx.y  = it1   [ 0, bs-1      ]
      // blockIdx.x  = it3   [ 0, iw-1      ]
      // threadIdx.x = it4   [ 0, 31        ]
      // icopy is supposed to be contiguous as it is a local temporary matrix
       
         
      int fout = (MAX(0,padtop-blockIdx.z)+stridey-1)/stridey;
      int fin = fout * stridey - padtop + blockIdx.z;

      if (fin < ih) 
      {
         inputptr += (blockIdx.y)*inputstr0+fin*inputstr1+(blockIdx.x)*inputstr2;
         icopyptr += blockIdx.z*bs*toh*tiw*ip+(blockIdx.y)*toh*tiw*ip+fout*tiw*ip+(padleft+blockIdx.x)*ip;
         
         int inputsize2   = ((ih-fin) + stridey - 1) / stridey;

         for (int it2=0; it2<inputsize2; it2++) { 
            for (int it4=threadIdx.x; it4<ip; it4+=blockDim.x) 
            {
               icopyptr[it4]=inputptr[it4];
            }
            inputptr += stridey*inputstr1;
            icopyptr += tiw*ip;
			}
      }
}





void unfoldInput(THCudaTensor* input, THCudaTensor* icopy, int stridey, int padtop, int padleft, int padright, int padbottom, int tih, int tiw, int toh, int tow)
{

  int bs = input->size[0];
  int ih = input->size[1];
  int iw = input->size[2];
  int ip = input->size[3];

  int inputstr0 = input->stride[0];
  int inputstr1 = input->stride[1];
  int inputstr2 = input->stride[2];
  int inputstr3 = input->stride[3];

  THCudaTensor_resize5d(icopy, stridey, bs, toh, tiw, ip);
  THCudaTensor_fill(icopy, 0);

  float* icopyptr=THCudaTensor_data(icopy);
  float* inputptr=THCudaTensor_data(input);

 
  if(ip<32 && THCudaTensor_isContiguous(input)) {
      dim3 icopyblocks(iw/(32/ip)+1, bs, stridey);
      dim3 icopythreads(MIN(32,ip), 32/ip);
      SCinputcopykernelsmall <<<icopyblocks, icopythreads>>> (inputptr, icopyptr, stridey, bs, ih, iw, ip, padtop, padleft, toh, tiw);
  }
  else {
      dim3 icopyblocks(iw, bs, stridey);
      dim3 icopythreads(32);
      SCinputcopykernel <<<icopyblocks, icopythreads>>> (inputptr, icopyptr, stridey, bs, ih, iw, ip, padtop, padleft, toh, tiw, inputstr0, inputstr1, inputstr2, inputstr3);
  }
  
}






__global__ void SCoutputcopykernel(float* outputptr, float* ocopyptr, float* biasptr, int bs, int oh, 
      int ow, int op, int toh, int tow, int outputstr0, int outputstr1, int outputstr2, int outputstr3)
      {
      /* blockIdx.z  = it1   [ 0, bs-1      ]
         blockIdx.y  = it2   [ 0, oh-1      ]
         blockIdx.x  = it3   [ 0, ow-1      ]
         threadIdx.x = it4   [ 0, 31        ]
       */      
         outputptr += (blockIdx.z)*outputstr0+(blockIdx.y)*outputstr1+(blockIdx.x)*outputstr2;
         ocopyptr  += (blockIdx.z)*toh*tow*op+(blockIdx.y)*tow*op+(blockIdx.x)*op;
         for (int it4=threadIdx.x; it4<op; it4+=blockDim.x) {
            outputptr[it4]=ocopyptr[it4] + biasptr[it4];
		   }
      }





void unpadAndAddBiasOutput(THCudaTensor* ocopy, THCudaTensor* output, THCudaTensor* bias, int bs, int oh, int ow, int op, int toh, int tow)
{
  // will output a contiguous matrix, except if the tensor is of proper size, in which case output will be recycled
  if(output->nDimension==4)
  {
     if(output->size[0] != bs || output->size[1] != oh || output->size[2] != ow || output->size[3] != op)
     {
         THCudaTensor_resize4d(output, bs, oh, ow, op);
     }
  }
  else //if the tensor doesn't exist...
  {
         THCudaTensor_resize4d(output, bs, oh, ow, op);
  }
   
  float* ocopyptr=THCudaTensor_data(ocopy);
  float* outputptr=THCudaTensor_data(output);
  float* biasptr=THCudaTensor_data(bias);

  dim3 ocopyblocks(ow, oh, bs);
  dim3 ocopythreads(32);

  int outputstr0 = output->stride[0];
  int outputstr1 = output->stride[1];
  int outputstr2 = output->stride[2];
  int outputstr3 = output->stride[3];
  
  SCoutputcopykernel <<<ocopyblocks, ocopythreads>>> (outputptr, ocopyptr, biasptr, bs, oh, ow, op, toh, tow, outputstr0, outputstr1, outputstr2, outputstr3);

}




static int cunxn_SpatialConvolution_updateOutput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *tmpweight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "tmpweight", "torch.CudaTensor");

  /* contiguity check */ 
  input = THCudaTensor_newContiguous(input);

  /* transpose weight dims 1 and 2 so it is in proper format */ 
	transposeWeightMatrix(tmpweight, weight);

  int stridex = luaT_getfieldcheckint(L, 1, "dW");
  int stridey = luaT_getfieldcheckint(L, 1, "dH");

  int padleft = luaT_getfieldcheckint(L, 1, "padleft");
  int padright = luaT_getfieldcheckint(L, 1, "padright");
  int padtop = luaT_getfieldcheckint(L, 1, "padtop");
  int padbottom = luaT_getfieldcheckint(L, 1, "padbottom");

  int overlap = luaT_getfieldcheckint(L, 1, "overlap");

  float onef=1;

  int bs = input->size[0];
  int ih = input->size[1];
  int iw = input->size[2];
  int ip = input->size[3];

  int inputstr0 = input->stride[0];
  int inputstr1 = input->stride[1];
  int inputstr2 = input->stride[2];
  int inputstr3 = input->stride[3];
  
  int kh = weight->size[0];
  int op = weight->size[1];
  int kw = weight->size[2];
//  printf("ip: %d, weight : %d\n", ip, weight->size[3]);
  assert(ip==weight->size[3]);
  
  /* compute output size */
  int ow = ( iw + padleft + padright - kw ) / stridex + 1;
  int oh = ( ih + padtop + padbottom - kh ) / stridey + 1;

  /* correct padright and padbottom */
  padright = ow * stridex + kw - stridex - iw - padleft;
  padbottom = oh * stridey + kh - stridey - ih - padtop;
  /* assert(not exact or padright ~= oldpadright, "horizontal size mismatch"); */
  /* assert(not exact or padbottom ~= oldpadbottom, "horizontal size mismatch"); */
  if (padright < 0)  { padright = 0;}
  if (padbottom < 0) { padbottom = 0;}

  /* input size with padding */
  int piw = padleft + iw + padright; 
  int pih = padtop + ih + padbottom;

  /* number of horizontal strides between nonoverlapping runs */
  int nxs = 1;
  if (!overlap) { nxs = (kw + stridex - 1) / stridex ;}

  /* total size of output buffer */
  int tow = (piw + stridex - 1) / stridex;
  int toh = (pih + stridey - 1) / stridey;

  /* total size of input and output buffers */
  int tiw = tow * stridex;
  int tih = toh * stridey;  
  assert(tiw >= piw && piw >= iw);
  assert(tih >= pih && pih >= ih);

  
	THCudaTensor* icopy = THCudaTensor_new();
	unfoldInput(input, icopy, stridey, padtop, padleft, padright, padbottom, tih, tiw, toh, tow);
  
  THCudaTensor* kcopy = weight;

  THCudaTensor* ocopy = THCudaTensor_newWithSize4d(bs, toh, tow, op);
  THCudaTensor_fill(ocopy, 0);

  hipblasHandle_t handle;
  hipblasStatus_t err = hipblasCreate(&handle);
  if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in creating handle"); }

   hipStream_t* streams = (hipStream_t*) malloc(nxs*sizeof(hipStream_t));



   /* call GEMM */
	int hcall;
   for (hcall=0; hcall<nxs; hcall++) {

		hipStreamCreate(&streams[hcall]);
		hipblasSetStream(handle, streams[hcall]);

	   int vcall;
      for (vcall=0; vcall<kh; vcall++) {
         int sq = vcall / stridey;
         int sr = vcall - sq * stridey;
         /* local icopy =  newSameTensor(input, stridey, bs, toh, tiw, ip) */
         /* float* iptr = torch.data(icopy[{sr+1,{},sq+1,hcall*stridex+1,{}}]) */
		   float* iptr = THCudaTensor_data(icopy);
		   iptr       += (sr)*icopy->stride[0] + (sq)*icopy->stride[2] +  (hcall*stridex)*icopy->stride[3];

         /* local kptr  = torch.data(kcopy:select(1,vcall+1)) */
		   float* kptr = THCudaTensor_data(kcopy);
		   kptr	 	+= vcall * kcopy->stride[0];

         /* local optr = torch.data(ocopy:select(3,hcall+1)) */
		   float* optr = THCudaTensor_data(ocopy);
         optr		+= hcall * ocopy->stride[2];


         int nrun = (bs-1)*toh*tow + oh*tow;
         int ngem = (nrun - hcall) / nxs;

         err = hipblasSgemm(handle,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           op, ngem, kw*ip,
                           &onef,
                           kptr, kw*ip,
                           iptr, nxs*stridex*ip,
                           &onef,
                           optr, nxs*op );     
              
              
              
         if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in sgemm"); }
      }
   }


   for (hcall=0; hcall<nxs; hcall++) {
		hipStreamDestroy(streams[hcall]);
	}

	free(streams);

  err = hipblasDestroy(handle);
  if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in destroying handle"); }

  unpadAndAddBiasOutput(ocopy, output, bias, bs, oh, ow, op, toh, tow);

  // check for errors
  hipError_t lasterror = hipGetLastError();
  if (lasterror != hipSuccess) {
    printf("error in SpatialConvolution.updateOutput: %s\n", hipGetErrorString(lasterror));
    THError("aborting");
  }
 
  // final cut:
  //THCudaTensor_free(input); 
  THCudaTensor_free(icopy);
  THCudaTensor_free(ocopy);
  //THCudaTensor_select(output, NULL, dimension, 0);

  return 1;
}







__global__ void SCkernelCopyReverse(float* weightptr, float* revkptr, int stridey, int stridex, int kouth, 
      int koutw, int kouto, int kouti, int sh, int so, int sw, int si, int kh, int kw, int ko, int ki)
{
   /*
         blockIdx.z  =    [ 0, ceil(ki/32)] -> usually this should be good
            inputplane = blockIdx.z * blockDim.x+threadIdx.x
         blockIdx.y  =    [ 0, stry-1    ]
         blockIdx.x  =    [ 0, strx-1    ]
         threadIdx.x =    [ 0, 31        ] -> weight input dim
         threadIdx.y =    [ 0, 31        ] -> weight output dim
            outputplane= iterator * blockDim.y + threadIdx.y
   */
   const int stry=blockIdx.y;
   const int strx=blockIdx.x;

   // put revkptr on proper stry,strx submatrix
   revkptr  +=    (blockIdx.y*stridex + blockIdx.x)*kouth*kouto*koutw*kouti;

   
   __shared__ float weightvalues[32][33];
   // for given x,y : weightvalues[inputplane][outputplane]
   
   
   int ith, itw, xcoord, ycoord, ito;
   for(ith=0; ith<kouth; ith++) {
      ycoord=kh-(ith*stridey+stry+1);
      if (ycoord<kh && ycoord>-1) {
         for(itw=0; itw<koutw; itw++) {
	   	   xcoord=kw-(itw*stridex+strx+1);
				if (xcoord<kw && xcoord>-1) {         

/*              int kh = weight->size[0];
              int op = weight->size[1];
              int kw = weight->size[2];
              assert(ip==weight->size[3]);            */
         
         for (ito=0; ito<(ko+blockDim.y-1)/blockDim.y; ito++) {

         /* iterate over tiles of size 32*32 */

         /* Step 1 : for a given (x,y)
            read weight(y, [32o], x, [32i]) and store the stuff in shmem */

                  const int curoplane=ito*blockDim.y+threadIdx.y;
                  const int curiplane=blockIdx.z * blockDim.x+threadIdx.x;
                  
                  if(curiplane<ki && curoplane<ko) {
                     weightvalues[threadIdx.x][threadIdx.y]=weightptr[ycoord*sh+xcoord*sw+(curoplane)*so+(curiplane)*si];
                  }
                  
                  __syncthreads();

         /* Step 2 : write revk(ith, [32i], itw, [32o]) in submatrix */
                  
                  const int reviplane=blockIdx.z * blockDim.y + threadIdx.y;
                  const int revoplane=ito*blockDim.x+threadIdx.x;
                  
                  if( reviplane < ki && revoplane < ko) {
                     revkptr[ith*kouto*koutw*kouti + itw*kouti + reviplane*koutw*kouti + revoplane] = weightvalues[threadIdx.y][threadIdx.x];
                  }

                  __syncthreads();

         }
         
         
   
         
         }
         }
      }
   }
   


   
}










__global__ void SCcopyGradOut(float* goptr, float* gocpyptr, int goh, int gow, int pgoh, int pgow, int revkh, int revkw, int op, int gradOutstr0, int gradOutstr1, int gradOutstr2, int gradOutstr3   )
{
   /* blockIdx.z  = [ 0, bs-1  ] (it1)
      blockIdx.y  = [ 0, goh-1 ] (it2)
      blockIdx.x  = [ 0, gow-1 ] (it3)
      threadIdx.x = [ 0, 31    ] (it4)
   */

   gocpyptr += ((blockIdx.z*pgoh+(revkh -1 + blockIdx.y))*pgow+(revkw-1+blockIdx.x))*op;
   goptr += ((blockIdx.z*goh+blockIdx.y)*gow+blockIdx.x)*gradOutstr2;

   int i;
   for(i=threadIdx.x; i<op; i+=blockDim.x)
   {
      gocpyptr[i]=goptr[i];
   }

}






__global__ void SCcopyGradinResult2(float* gradinptr, float* resptr, int throwawayx, int throwawayy, int stridey, int rs0, int rs1, int rs2, int gs0, int gs1, int gs2, int gs3, int ip, int gih, int padtop, int padleft, int ih, int iw)
{
   /*
      blockIdx.z  = [ 0, bs-1 ] (it1)
      blockIdx.y  = [ 0 ] 
      blockIdx.x  = [ 0, iw ] (it3) (giw-throwawayx ?)
      threadIdx.x = [ 0, 31   ] (it4)
   */

   int itres0 = 0;
   int itgi0  = 0;

   int starty, sizey;

   for(int stry=stridey; stry>0; stry--) 
   {
   	int throwaway = stridey-stry < throwawayy;
	   if(throwaway) {
		   starty = (stridey-stry+1) - throwawayy + stridey -1 ;
		   sizey  = gih-1;
    	}
	   else 	{ 
		   starty = (stridey-stry+1) - throwawayy -1 ;
		   sizey  = gih;
	   }

	   itgi0 = (stry-1)*gs0;
	
	   int itres1 = itres0 + blockIdx.z*rs0;
	   int itgi1  = itgi0  + blockIdx.z*gs1;
	   int it2, it3, it4;
	   for (it2=0; it2<sizey; it2++) { 
		   int itres2 = itres1 + (starty + it2*stridey)*rs1;
		   int itgi2  = itgi1 + it2*gs2;
		   if(throwaway) {itgi2 += gs2;}
			   int itres3 = itres2 + blockIdx.x*rs2;
			   int itgi3  = itgi2 + (throwawayx+blockIdx.x)*gs3;
            for(int it4=threadIdx.x; it4<ip; it4+=blockDim.x)
            {
               resptr[itres3+it4]= gradinptr[itgi3+it4];
            }
		} 
	}
}




void copyKernelReverse(THCudaTensor* weight, THCudaTensor* revk, int stridey, int stridex, int kh, int kw)
{
	/* the point here is to flip the kernels and stripe them */ 


   int ko = weight->size[1];
   int ki = weight->size[3];
   
   int sh = weight->stride[0];
   int so = weight->stride[1];
   int sw = weight->stride[2];
   int si = weight->stride[3];
   
   
   
   int kouth=(kh+stridey-1)/stridey;
   int kouto=ki;
   int koutw=(kw+stridex-1)/stridex;
   int kouti=ko;

   /* clean this after... */
   int revkh=kouth;
   int revkw=koutw;

   THLongStorage *revksize = THLongStorage_newWithSize(6);
   revksize->data[0]=stridey;
   revksize->data[1]=stridex;
   revksize->data[2]=kouth;
   revksize->data[3]=kouto;
   revksize->data[4]=koutw;
   revksize->data[5]=kouti;

   THCudaTensor_resize(revk, revksize, NULL);
   THCudaTensor_fill(revk, 0);
   
   float* weightptr=THCudaTensor_data(weight);
   float* revkptr=THCudaTensor_data(revk);
   
   dim3 kcrblocks(stridex, stridey, (ki+31)/32);
   dim3 kcrthreads(32,32);
   
   
   SCkernelCopyReverse <<<kcrblocks, kcrthreads>>>(weightptr, revkptr, stridey, stridex, kouth, 
      koutw, kouto, kouti, sh, so, sw, si, kh, kw, ko, ki);
   /*
         blockIdx.z  =    [ 0, ceil(ki/32)] -> parallelizing over inputplanes dimension : 
            usually there will be lots of them except in data layer where there is no backprop
            inputplane = blockIdx.z * blockDim.x+threadIdx.x
         blockIdx.y  =    [ 0, stry-1    ]
         blockIdx.x  =    [ 0, strx-1    ]
         threadIdx.x =    [ 0, 31        ] -> weight input dim
         threadIdx.y =    [ 0, 31        ] -> weight output dim
            outputplane= iterator * blockDim.y + threadIdx.y
   */
   
   
   /* end of copyKernelReverse */

}






static int cunxn_SpatialConvolution_updateGradInput(lua_State *L)
{
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "tmpweight", "torch.CudaTensor");
  THCudaTensor *tmpweight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInput  = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *result = THCudaTensor_new();
  THCudaTensor *revk;

  /* contiguity check */ 
  input = THCudaTensor_newContiguous(input);
  gradOutput = THCudaTensor_newContiguous(gradOutput);



  /* transpose weight dims 1 and 2 so it is in proper format */ 
	transposeWeightMatrix(tmpweight, weight);

  int stridex = luaT_getfieldcheckint(L, 1, "dW");
  int stridey = luaT_getfieldcheckint(L, 1, "dH");

  int padleft = luaT_getfieldcheckint(L, 1, "padleft");
  int padright = luaT_getfieldcheckint(L, 1, "padright");
  int padtop = luaT_getfieldcheckint(L, 1, "padtop");
  int padbottom = luaT_getfieldcheckint(L, 1, "padbottom");

  int overlap = luaT_getfieldcheckint(L, 1, "overlap");

  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");

  int bs = input->size[0];
  int ih = input->size[1];
  int iw = input->size[2];
  int ip = input->size[3];

  int kh = weight->size[0];
  int op = weight->size[1];
  int kw = weight->size[2];
  assert(ip==weight->size[3]);


   assert(gradOutput->nDimension == 4);
   assert(bs == gradOutput->size[0]);
   /* check that output h,w sizes match gradOutput sizes      */
   int goh = gradOutput->size[1];
   int gow = gradOutput->size[2];
   assert(goh == (ih + padtop + padbottom - kh) / stridey + 1) ;
   assert(gow == (iw + padleft + padright - kw) / stridex + 1) ;
   assert(op == gradOutput->size[3]);



   /*copyKernelReverse*/
	revk = THCudaTensor_new();
   copyKernelReverse(weight, revk, stridey, stridex, kh, kw);

   /* create gradinput tensor :*/
   int revkh=(kh+stridey-1)/stridey;
   int revkw=(kw+stridex-1)/stridex;
   int giw = ( gow + revkw -1 ) * stridex;
   int gih = ( goh + revkh -1 ) ;

   THLongStorage *gradinsize = THLongStorage_newWithSize(5);
   gradinsize->data[0]=stridey;
   gradinsize->data[1]=bs;
   gradinsize->data[2]=gih;
   gradinsize->data[3]=giw;
   gradinsize->data[4]=ip;

   THCudaTensor * gradin = THCudaTensor_newWithSize(gradinsize, NULL);
   THCudaTensor_fill(gradin, 0);
   


	/* create gradOutput buffer tensor */
   /* pad gradoutput tensor :*/
   int pgow = ( gow + revkw -1 );
   int pgoh = ( goh + revkh -1 );
   /* here we take bs+1 to have some zero-padding at the end of the matrix */
   /* it only costs some memory. GEMM does not use it. */
   THLongStorage *gradoutsize = THLongStorage_newWithSize(4);
   gradoutsize->data[0]=bs+1;
   gradoutsize->data[1]=pgoh;
   gradoutsize->data[2]=pgow;
   gradoutsize->data[3]=op;

   THCudaTensor * gradOutCopy = THCudaTensor_newWithSize(gradoutsize, NULL);
   THCudaTensor_fill(gradOutCopy, 0);

   float* goptr=THCudaTensor_data(gradOutput);
   float* gocpyptr=THCudaTensor_data(gradOutCopy);

   dim3 cgoblocks(gow, goh, bs);
   dim3 cgothreads(32);
   
   int gradOutstr0 = gradOutput->stride[0];
   int gradOutstr1 = gradOutput->stride[1];
   int gradOutstr2 = gradOutput->stride[2];
   int gradOutstr3 = gradOutput->stride[3];
   
   SCcopyGradOut <<< cgoblocks, cgothreads >>>(goptr, gocpyptr, goh, gow, pgoh, pgow, revkh, revkw, op, gradOutstr0, gradOutstr1, gradOutstr2, gradOutstr3);


   
   float onef=1;
   
  hipblasHandle_t handle;
  hipblasStatus_t err = hipblasCreate(&handle);
  if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in creating handle"); }

   
   /* GEMM calls : */
	int nxs=1;
	if(!overlap) {
	   nxs=revkw; 
	   //printf("no overlap");
	}

   hipStream_t* streams = (hipStream_t*) malloc(nxs*stridex*stridey*sizeof(hipStream_t));

	for (int hcall=0; hcall<nxs; hcall++) {
	   for (int stry=0; stry<stridey; stry++) {
		   for (int strx=0; strx<stridex; strx++) {
				int idx=hcall*stridex*stridey+stry*stridex+strx;
				hipStreamCreate(&streams[idx]);
				hipblasSetStream(handle, streams[idx]);


			   for (int vcall=0; vcall<revkh; vcall++) {
				   float* gradoutptr  = THCudaTensor_data(gradOutCopy);
				   gradoutptr		   += (revkh-vcall-1)*gradOutCopy->stride[1] + hcall*gradOutCopy->stride[2];
               int ldgradout      = op*nxs;
                     
				   float* krevptr	    = THCudaTensor_data(revk);
				   krevptr 		      += (stry)*revk->stride[0] + (strx)*revk->stride[1] + (revkh-vcall-1)*revk->stride[2];
               int szkrev         = op*revkw;
               int ldkrev     	 = op*revkw;
                  
				   float* gradinptr	 = THCudaTensor_data(gradin);
				   gradinptr		+= (stry)*gradin->stride[0] + (stridex-(strx)-1+hcall*stridex)*gradin->stride[3];
               int ldgradin   	 = ip * stridex * nxs;
                  
               int nspots         = giw/stridex*gih*bs;
               int ngem           = (nspots-hcall+nxs-1)/nxs;
                  
               err = hipblasSgemm(handle,
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           ip, ngem, szkrev,
                           &onef,
                           krevptr, ldkrev,
                           gradoutptr, ldgradout,
                           &onef,
                           gradinptr, ldgradin );

               if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in sgemm"); }
			   }
		   }
	   }
   }

   for (int idx=0; idx<nxs*stridex*stridey; idx++) {
		hipStreamDestroy(streams[idx]);
	}

	free(streams);

  err = hipblasDestroy(handle);
  if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in destroying handle"); }
   
   
   
   
   
   
   
   
   
     /* correct padright and padbottom */
   padright = gow * stridex + kw - stridex - iw - padleft;
   padbottom = goh * stridey + kh - stridey - ih - padtop;
   if (padright < 0)  { padright = 0;}
   if (padbottom < 0) { padbottom = 0;}

   /* clean this after */ 
   int resw=padleft + iw + padright;
   int resh=padtop + ih + padbottom;
    
   int throwawayx=stridex - kw%stridex;
   int throwawayy=stridey - kh%stridey;
   if (stridex==1 || stridex==throwawayx) { throwawayx=0 ; } 
   if (stridey==1 || stridey==throwawayy) { throwawayy=0 ; }

   THCudaTensor_resize4d(result, bs, resh, resw, ip);
   THCudaTensor_fill(result, 0);


   float* gradinptr = THCudaTensor_data(gradin);
   float* resptr = THCudaTensor_data(result);

   int rs0 = result->stride[0];
   int rs1 = result->stride[1];
   int rs2 = result->stride[2];
   int gs0 = gradin->stride[0]; 
   int gs1 = gradin->stride[1];
   int gs2 = gradin->stride[2];
   int gs3 = gradin->stride[3];
 
   dim3 cgirblocks2(giw-throwawayx, 1, bs);
   dim3 cgirthreads2(32);

   SCcopyGradinResult2 <<<cgirblocks2,cgirthreads2>>> (gradinptr, resptr, throwawayx, throwawayy, stridey, rs0, rs1, rs2, gs0, gs1, gs2, gs3, ip, gih, padtop, padleft, ih, iw);

	THCudaTensor_narrow(result, NULL, 1, padtop, ih);
	THCudaTensor_narrow(result, NULL, 2, padleft, iw);

	THCudaTensor_resizeAs(gradInput, result);
	THCudaTensor_freeCopyTo(result, gradInput);

   hipDeviceSynchronize();

   THCudaTensor_free(gradin);
   THCudaTensor_free(revk);
   THCudaTensor_free(gradOutCopy);
   
   
   

  // check for errors
  hipError_t err2 = hipGetLastError();
  if (err2 != hipSuccess) {
    printf("error in SpatialConvolution.updateOutput: %s\n", hipGetErrorString(err2));
    THError("aborting");
  }

  return 1;
}





__global__ void SCcopyGradOutInBuffer(float* goptr, float* gocpyptr, int oh, int ow, int toh, int tow, int op, int gradOutstr0, int gradOutstr1, int gradOutstr2, int gradOutstr3)
{
   /* blockIdx.z  = [ 0, bs-1  ] (it1)
      blockIdx.y  = [ 0, oh-1  ] (it2)
      blockIdx.x  = [ 0, ow-1  ] (it3)
      threadIdx.x = [ 0, 31    ] (it4)
   */

   gocpyptr += blockIdx.z*toh*tow*op + blockIdx.y*tow*op + blockIdx.x*op;
   goptr += ((blockIdx.z*oh+blockIdx.y)*ow+blockIdx.x)*gradOutstr2;

   int i;
   for(i=threadIdx.x; i<op; i+=blockDim.x)
   {
      gocpyptr[i]=goptr[i];
   }

}




__global__ void SCcomputeGradBias(float* goptr, float* gradbiasptr, int bs, int oh, int ow, int op, float scale, int gradOutstr0, int gradOutstr1, int gradOutstr2, int gradOutstr3)
{
   /* blockIdx.x  = [ 0, ceil(op/32) ]
      blockIdx.y  = [ 0, bs-1        ]
      threadIdx.x = [ 0, 31          ]   
   */

   goptr += blockIdx.y*gradOutstr0;
   const int idx = blockIdx.x * blockDim.x + threadIdx.x;
   
   float b=0;
   
   if (idx<op) {
      for(int i=0; i<oh*ow; i++) {
         b += goptr[i*gradOutstr2 + idx];
      }
   atomicAdd(&gradbiasptr[idx], b*scale);
   }
   
}






static int cunxn_SpatialConvolution_accGradParameters(lua_State *L)
{



  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "tmpgradweight", "torch.CudaTensor");
  THCudaTensor *tmpgradweight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");

  /* contiguity check */ 
  input = THCudaTensor_newContiguous(input);
  gradOutput = THCudaTensor_newContiguous(gradOutput);


  /* transpose weight dims 1 and 2 so it is in proper format */

	transposeWeightMatrix(tmpgradweight, gradWeight);


  float scale = luaL_optnumber(L, 4, 1);

  int stridex = luaT_getfieldcheckint(L, 1, "dW");
  int stridey = luaT_getfieldcheckint(L, 1, "dH");

  int padleft = luaT_getfieldcheckint(L, 1, "padleft");
  int padright = luaT_getfieldcheckint(L, 1, "padright");
  int padtop = luaT_getfieldcheckint(L, 1, "padtop");
  int padbottom = luaT_getfieldcheckint(L, 1, "padbottom");

  int overlap = luaT_getfieldcheckint(L, 1, "overlap");


  float onef=1;


  int bs = input->size[0];
  int ih = input->size[1];
  int iw = input->size[2];
  int ip = input->size[3];

  int inputstr0 = input->stride[0];
  int inputstr1 = input->stride[1];
  int inputstr2 = input->stride[2];
  int inputstr3 = input->stride[3];
  
  int kh = gradWeight->size[0];
  int op = gradWeight->size[1];
  int kw = gradWeight->size[2];
  assert(ip==gradWeight->size[3]);
  
  /* compute output size */
  int ow = ( iw + padleft + padright - kw ) / stridex + 1;
  int oh = ( ih + padtop + padbottom - kh ) / stridey + 1;

  /* correct padright and padbottom */
//  int oldpadright = padright;
//  int oldpadbottom = padbottom;
  padright = ow * stridex + kw - stridex - iw - padleft;
  padbottom = oh * stridey + kh - stridey - ih - padtop;
  /* assert(not exact or padright ~= oldpadright, "horizontal size mismatch"); */
  /* assert(not exact or padbottom ~= oldpadbottom, "horizontal size mismatch"); */
  if (padright < 0)  { padright = 0;}
  if (padbottom < 0) { padbottom = 0;}

  /* input size with padding */
  int piw = padleft + iw + padright; 
  int pih = padtop + ih + padbottom;

  /* number of horizontal strides between nonoverlapping runs */
  int nxs = 1;
  if (!overlap) { nxs = (kw + stridex - 1) / stridex ;}

  /* total size of output buffer */
  int tow = (piw + stridex - 1) / stridex;
  int toh = (pih + stridey - 1) / stridey;

  /* total size of input and output buffers */
  int tiw = tow * stridex;
  int tih = toh * stridey;  
  assert(tiw >= piw && piw >= iw);
  assert(tih >= pih && pih >= ih);

  /*icopy =  newSameTensor(input, stridey, bs, toh, tiw, ip) */
  THLongStorage *icopysize = THLongStorage_newWithSize(5);
  icopysize->data[0]=stridey;
  icopysize->data[1]=bs;
  icopysize->data[2]=toh;
  icopysize->data[3]=tiw;
  icopysize->data[4]=ip;
  THCudaTensor* icopy = THCudaTensor_newWithSize(icopysize, NULL);
  THCudaTensor_fill(icopy, 0);


  float* icopyptr=THCudaTensor_data(icopy);
  float* inputptr=THCudaTensor_data(input);

 
  if(ip<32 && THCudaTensor_isContiguous(input)) {
      dim3 icopyblocks(iw/(32/ip)+1, bs, stridey);
      dim3 icopythreads(MIN(32,ip), 32/ip);
      SCinputcopykernelsmall <<<icopyblocks, icopythreads>>> (inputptr, icopyptr, stridey, bs, ih, iw, ip, padtop, padleft, toh, tiw);
  }
  else {
      dim3 icopyblocks(iw, bs, stridey);
      dim3 icopythreads(32);
      SCinputcopykernel <<<icopyblocks, icopythreads>>> (inputptr, icopyptr, stridey, bs, ih, iw, ip, padtop, padleft, toh, tiw, inputstr0, inputstr1, inputstr2, inputstr3);
  }
  

  THCudaTensor* kcopy = gradWeight;
  THCudaTensor* ocopy = THCudaTensor_newWithSize4d(bs, toh, tow, op);
  THCudaTensor_fill(ocopy, 0);
  
  float* gradoutptr=THCudaTensor_data(gradOutput);
  float* ocpyptr=THCudaTensor_data(ocopy);
  
  dim3 goibblocks(ow, oh, bs);
  dim3 goibthreads(32);

  int gradOutstr0 = gradOutput->stride[0];
  int gradOutstr1 = gradOutput->stride[1];
  int gradOutstr2 = gradOutput->stride[2];
  int gradOutstr3 = gradOutput->stride[3];
  
  
   SCcopyGradOutInBuffer <<<goibblocks,goibthreads>>>(gradoutptr, ocpyptr, oh, ow, toh, tow, op, gradOutstr0, gradOutstr1, gradOutstr2, gradOutstr3);

   /* blockIdx.z  = [ 0, bs-1  ] (it1)
      blockIdx.y  = [ 0, oh-1  ] (it2)
      blockIdx.x  = [ 0, ow-1  ] (it3)
      threadIdx.x = [ 0, 31    ] (it4)
   */


  float* gradbiasptr=THCudaTensor_data(gradBias);
  
  dim3 gbblocks((op+31)/32, bs);
  dim3 gbthreads(32);
  SCcomputeGradBias <<< gbblocks, gbthreads >>> (gradoutptr, gradbiasptr, bs, oh, ow, op, scale, gradOutstr0, gradOutstr1, gradOutstr2, gradOutstr3);
  
   /* blockIdx.x  = [ 0, ceil(op/32) ]
      threadIdx.x = [ 0, 31          ]   
   */


  hipblasHandle_t handle;
  hipblasStatus_t err = hipblasCreate(&handle);
  if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in creating handle"); }

   hipStream_t* streams = (hipStream_t*) malloc(kh*sizeof(hipStream_t));
   for (int idx=0; idx<kh; idx++) {
		hipStreamCreate(&streams[idx]);
	}

   /* call GEMM */
	int hcall;
   for (hcall=0; hcall<nxs; hcall++) {
	   int vcall;
      for (vcall=0; vcall<kh; vcall++) {
			hipblasSetStream(handle, streams[vcall]);
         int sq = vcall / stridey;
         int sr = vcall - sq * stridey;
         /* local icopy =  newSameTensor(input, stridey, bs, toh, tiw, ip) */
         /* float* iptr = torch.data(icopy[{sr+1,{},sq+1,hcall*stridex+1,{}}]) */
		   float* iptr = THCudaTensor_data(icopy);
		   iptr       += (sr)*icopy->stride[0] + (sq)*icopy->stride[2] +  (hcall*stridex)*icopy->stride[3];

         /* local kptr  = torch.data(kcopy:select(1,vcall+1)) */
		   float* kptr = THCudaTensor_data(kcopy);
		   kptr	 	+= vcall * kcopy->stride[0];

         /* local optr = torch.data(ocopy:select(3,hcall+1)) */
		   float* optr = THCudaTensor_data(ocopy);
         optr		+= hcall * ocopy->stride[2];


         int nrun = (bs-1)*toh*tow + oh*tow;
         int ngem = (nrun - hcall) / nxs;

         //printf("calling sgemm...");

         /*THBlas_(gemm)('T','N', op, ngem, kw*ip, 
              1, kptr, kw*ip, iptr, nxs*stridex*ip,
              1, optr, nxs*op ); */
         err = hipblasSgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_T,
                           kw*ip,op, ngem, 
                           &scale,
                           iptr, nxs*stridex*ip, 
                           optr, nxs*op, 
                           &onef,
                           kptr, kw*ip );     
              
              
              
         if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in sgemm"); }
         //else {printf("called sgemm..."); }
      }
   }

   for (int idx=0; idx<kh; idx++) {
		hipStreamDestroy(streams[idx]);
	}

	free(streams);

  err = hipblasDestroy(handle);
  if (err != HIPBLAS_STATUS_SUCCESS) { printf("error in destroying handle"); }





  /* transpose weight dims 1 and 2 so it is in proper format */
	transposeWeightMatrix(gradWeight, tmpgradweight);


  // check for errors
  hipError_t lasterror = hipGetLastError();
  if (lasterror != hipSuccess) {
    printf("error in SpatialConvolution.updateOutput: %s\n", hipGetErrorString(lasterror));
    THError("aborting");
  }
 
  // final cut:
  //THCudaTensor_free(input); 
  THCudaTensor_free(icopy);
  THCudaTensor_free(ocopy);
  //THCudaTensor_select(output, NULL, dimension, 0);

  return 1;

}



__global__ void SCclipWeightsKernel(float* wdataptr, float normbound, int kh, int op, int kw, int ip, int str0, int str1)
{
   /* blockIdx.x  = [ 0, op    ] ()
      threadIdx.x = [ 0, 31    ] ()
   */

   wdataptr += blockIdx.x*str1;

   volatile __shared__ float sqrsums[32];
   int ith, it, i;
   float sqrsum=0;
   float current;
   int numelperline=kw*ip;
   for (ith=0; ith<kh; ith++)
   {
      for(i=threadIdx.x; i<numelperline; i+=blockDim.x)
      {
         current=wdataptr[ith*str0+i];
         sqrsum+=current*current;
      }
   }

   sqrsums[threadIdx.x]=sqrsum;
   
   // NVCC : Y U NO __SHFL ?
   if (threadIdx.x < 16)
   {
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 16];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 8];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 4];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 2];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 1];
      sqrsums[threadIdx.x + 1] = sqrsums[threadIdx.x];
      sqrsums[threadIdx.x + 2] = sqrsums[threadIdx.x];
      sqrsums[threadIdx.x + 4] = sqrsums[threadIdx.x];
      sqrsums[threadIdx.x + 8] = sqrsums[threadIdx.x];
      sqrsums[threadIdx.x + 16] = sqrsums[threadIdx.x];
   }

   sqrsum=sqrsums[threadIdx.x];   


   // replace with this when __shfl works :
   /*if (threadIdx.x < 16)
   {
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 16];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 8];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 4];
      sqrsums[threadIdx.x] += sqrsums[threadIdx.x + 2];
   }
   if (threadIdx.x == 0)
   {
      sqrsum = sqrsums[0]+sqrsums[1];
   }
   
   sqrsum = __shfl(sqrsum, 0);*/
   
   if(sqrsum>normbound*normbound)
   {
      float scale = normbound/sqrt(sqrsum); 
      for (ith=0; ith<kh; ith++)
      {
         for(i=threadIdx.x; i<numelperline; i+=blockDim.x)
         {
            wdataptr[ith*str0+i] *= scale;
            //wdataptr[ith*str0+i] =0; // for testing...
         }
      }
   }
}





static int cunxn_SpatialConvolution_clipWeights(lua_State *L)
{
  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  float normbound = luaL_optnumber(L, 2, 1);

  int kh = weight->size[0];
  int op = weight->size[1];
  int kw = weight->size[2];
  int ip = weight->size[3];
  
  int str0 = weight->stride[0];
  int str1 = weight->stride[1];
  int str2 = weight->stride[2];
  int str3 = weight->stride[3];

  float* wdata=THCudaTensor_data(weight);

  dim3 blocks(op);
  dim3 threads(32);
  
  SCclipWeightsKernel <<<blocks, threads>>>(wdata, normbound, kh, op, kw, ip, str0, str1);

  return 1;
}








static const struct luaL_Reg cunxn_SpatialConvolution__ [] = {
  {"SpatialConvolution_updateOutput", cunxn_SpatialConvolution_updateOutput},
  {"SpatialConvolution_updateGradInput", cunxn_SpatialConvolution_updateGradInput},
  {"SpatialConvolution_accGradParameters", cunxn_SpatialConvolution_accGradParameters},
  {"SpatialConvolution_clipWeights", cunxn_SpatialConvolution_clipWeights},
  {NULL, NULL}
};

static void cunxn_SpatialConvolution_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunxn_SpatialConvolution__, "nxn");
  lua_pop(L,1);
}

